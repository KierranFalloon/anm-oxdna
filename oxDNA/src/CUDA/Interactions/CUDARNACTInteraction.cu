#include "hip/hip_runtime.h"
/*
 * CUDARNACTInteraction.cu
 *
 *  Created on: 25 Aug 2020
 *      Author: jonah
 */

#include "CUDARNACTInteraction.h"
#include "CUDARNAInteraction.h" //questionable
#include "CUDA_RNA.cuh"
#include "CUDA_ACT_R.cuh"
#include "../Lists/CUDASimpleVerletList.h"
#include "../Lists/CUDANoList.h"
#include "../../Interactions/RNAInteraction2.h"
#include "../../Interactions/RNACTInteraction.h"

void copy_Model_to_CUDAModel_RNACT(Model& model_from, CUDAModel& model_to)
{
    model_to.RNA_POS_BACK =  model_from.RNA_POS_BACK ;
    model_to.RNA_POS_STACK =  model_from.RNA_POS_STACK ;
    model_to.RNA_POS_BASE =  model_from.RNA_POS_BASE ;
    model_to.RNA_GAMMA =  model_from.RNA_GAMMA ;
    model_to.RNA_POS_STACK_3_a1 =  model_from.RNA_POS_STACK_3_a1 ;
    model_to.RNA_POS_STACK_3_a2 =  model_from.RNA_POS_STACK_3_a2 ;
    model_to.RNA_POS_STACK_5_a1 =  model_from.RNA_POS_STACK_5_a1 ;
    model_to.RNA_POS_STACK_5_a2 =  model_from.RNA_POS_STACK_5_a2 ;
    model_to.RNA_FENE_EPS =  model_from.RNA_FENE_EPS ;
    model_to.RNA_FENE_R0 =  model_from.RNA_FENE_R0 ;
    model_to.RNA_FENE_DELTA =  model_from.RNA_FENE_DELTA ;
    model_to.RNA_FENE_DELTA2 =  model_from.RNA_FENE_DELTA2 ;
    model_to.RNA_EXCL_EPS =  model_from.RNA_EXCL_EPS ;
    model_to.RNA_EXCL_S1 =  model_from.RNA_EXCL_S1 ;
    model_to.RNA_EXCL_S2 =  model_from.RNA_EXCL_S2 ;
    model_to.RNA_EXCL_S3 =  model_from.RNA_EXCL_S3 ;
    model_to.RNA_EXCL_S4 =  model_from.RNA_EXCL_S4 ;
    model_to.RNA_EXCL_R1 =  model_from.RNA_EXCL_R1 ;
    model_to.RNA_EXCL_R2 =  model_from.RNA_EXCL_R2 ;
    model_to.RNA_EXCL_R3 =  model_from.RNA_EXCL_R3 ;
    model_to.RNA_EXCL_R4 =  model_from.RNA_EXCL_R4 ;
    model_to.RNA_EXCL_B1 =  model_from.RNA_EXCL_B1 ;
    model_to.RNA_EXCL_B2 =  model_from.RNA_EXCL_B2 ;
    model_to.RNA_EXCL_B3 =  model_from.RNA_EXCL_B3 ;
    model_to.RNA_EXCL_B4 =  model_from.RNA_EXCL_B4 ;
    model_to.RNA_EXCL_RC1 =  model_from.RNA_EXCL_RC1 ;
    model_to.RNA_EXCL_RC2 =  model_from.RNA_EXCL_RC2 ;
    model_to.RNA_EXCL_RC3 =  model_from.RNA_EXCL_RC3 ;
    model_to.RNA_EXCL_RC4 =  model_from.RNA_EXCL_RC4 ;
    model_to.RNA_HYDR_EPS =  model_from.RNA_HYDR_EPS ;
    model_to.RNA_HYDR_A =  model_from.RNA_HYDR_A ;
    model_to.RNA_HYDR_RC =  model_from.RNA_HYDR_RC ;
    model_to.RNA_HYDR_R0 =  model_from.RNA_HYDR_R0 ;
    model_to.RNA_HYDR_BLOW =  model_from.RNA_HYDR_BLOW ;
    model_to.RNA_HYDR_BHIGH =  model_from.RNA_HYDR_BHIGH ;
    model_to.RNA_HYDR_RLOW =  model_from.RNA_HYDR_RLOW ;
    model_to.RNA_HYDR_RHIGH =  model_from.RNA_HYDR_RHIGH ;
    model_to.RNA_HYDR_RCLOW =  model_from.RNA_HYDR_RCLOW ;
    model_to.RNA_HYDR_RCHIGH =  model_from.RNA_HYDR_RCHIGH ;
    model_to.RNA_HYDR_THETA1_A =  model_from.RNA_HYDR_THETA1_A ;
    model_to.RNA_HYDR_THETA1_B =  model_from.RNA_HYDR_THETA1_B ;
    model_to.RNA_HYDR_THETA1_T0 =  model_from.RNA_HYDR_THETA1_T0 ;
    model_to.RNA_HYDR_THETA1_TS =  model_from.RNA_HYDR_THETA1_TS ;
    model_to.RNA_HYDR_THETA1_TC =  model_from.RNA_HYDR_THETA1_TC ;
    model_to.RNA_HYDR_THETA2_A =  model_from.RNA_HYDR_THETA2_A ;
    model_to.RNA_HYDR_THETA2_B =  model_from.RNA_HYDR_THETA2_B ;
    model_to.RNA_HYDR_THETA2_T0 =  model_from.RNA_HYDR_THETA2_T0 ;
    model_to.RNA_HYDR_THETA2_TS =  model_from.RNA_HYDR_THETA2_TS ;
    model_to.RNA_HYDR_THETA2_TC =  model_from.RNA_HYDR_THETA2_TC ;
    model_to.RNA_HYDR_THETA3_A =  model_from.RNA_HYDR_THETA3_A ;
    model_to.RNA_HYDR_THETA3_B =  model_from.RNA_HYDR_THETA3_B ;
    model_to.RNA_HYDR_THETA3_T0 =  model_from.RNA_HYDR_THETA3_T0 ;
    model_to.RNA_HYDR_THETA3_TS =  model_from.RNA_HYDR_THETA3_TS ;
    model_to.RNA_HYDR_THETA3_TC =  model_from.RNA_HYDR_THETA3_TC ;
    model_to.RNA_HYDR_THETA4_A =  model_from.RNA_HYDR_THETA4_A ;
    model_to.RNA_HYDR_THETA4_B =  model_from.RNA_HYDR_THETA4_B ;
    model_to.RNA_HYDR_THETA4_T0 =  model_from.RNA_HYDR_THETA4_T0 ;
    model_to.RNA_HYDR_THETA4_TS =  model_from.RNA_HYDR_THETA4_TS ;
    model_to.RNA_HYDR_THETA4_TC =  model_from.RNA_HYDR_THETA4_TC ;
    model_to.RNA_HYDR_THETA7_A =  model_from.RNA_HYDR_THETA7_A ;
    model_to.RNA_HYDR_THETA7_B =  model_from.RNA_HYDR_THETA7_B ;
    model_to.RNA_HYDR_THETA7_T0 =  model_from.RNA_HYDR_THETA7_T0 ;
    model_to.RNA_HYDR_THETA7_TS =  model_from.RNA_HYDR_THETA7_TS ;
    model_to.RNA_HYDR_THETA7_TC =  model_from.RNA_HYDR_THETA7_TC ;
    model_to.RNA_HYDR_THETA8_A =  model_from.RNA_HYDR_THETA8_A ;
    model_to.RNA_HYDR_THETA8_B =  model_from.RNA_HYDR_THETA8_B ;
    model_to.RNA_HYDR_THETA8_T0 =  model_from.RNA_HYDR_THETA8_T0 ;
    model_to.RNA_HYDR_THETA8_TS =  model_from.RNA_HYDR_THETA8_TS ;
    model_to.RNA_HYDR_THETA8_TC =  model_from.RNA_HYDR_THETA8_TC ;
    model_to.RNA_STCK_BASE_EPS =  model_from.RNA_STCK_BASE_EPS ;
    model_to.RNA_STCK_FACT_EPS =  model_from.RNA_STCK_FACT_EPS ;
    model_to.RNA_STCK_A =  model_from.RNA_STCK_A ;
    model_to.RNA_STCK_RC =  model_from.RNA_STCK_RC ;
    model_to.RNA_STCK_R0 =  model_from.RNA_STCK_R0 ;
    model_to.RNA_STCK_BLOW =  model_from.RNA_STCK_BLOW ;
    model_to.RNA_STCK_BHIGH =  model_from.RNA_STCK_BHIGH ;
    model_to.RNA_STCK_RLOW =  model_from.RNA_STCK_RLOW ;
    model_to.RNA_STCK_RHIGH =  model_from.RNA_STCK_RHIGH ;
    model_to.RNA_STCK_RCLOW =  model_from.RNA_STCK_RCLOW ;
    model_to.RNA_STCK_RCHIGH =  model_from.RNA_STCK_RCHIGH ;
    model_to.RNA_STCK_THETA4_A =  model_from.RNA_STCK_THETA4_A ;
    model_to.RNA_STCK_THETA4_B =  model_from.RNA_STCK_THETA4_B ;
    model_to.RNA_STCK_THETA4_T0 =  model_from.RNA_STCK_THETA4_T0 ;
    model_to.RNA_STCK_THETA4_TS =  model_from.RNA_STCK_THETA4_TS ;
    model_to.RNA_STCK_THETA4_TC =  model_from.RNA_STCK_THETA4_TC ;
    model_to.RNA_STCK_THETA5_A =  model_from.RNA_STCK_THETA5_A ;
    model_to.RNA_STCK_THETA5_B =  model_from.RNA_STCK_THETA5_B ;
    model_to.RNA_STCK_THETA5_T0 =  model_from.RNA_STCK_THETA5_T0 ;
    model_to.RNA_STCK_THETA5_TS =  model_from.RNA_STCK_THETA5_TS ;
    model_to.RNA_STCK_THETA5_TC =  model_from.RNA_STCK_THETA5_TC ;
    model_to.RNA_STCK_THETA6_A =  model_from.RNA_STCK_THETA6_A ;
    model_to.RNA_STCK_THETA6_B =  model_from.RNA_STCK_THETA6_B ;
    model_to.RNA_STCK_THETA6_T0 =  model_from.RNA_STCK_THETA6_T0 ;
    model_to.RNA_STCK_THETA6_TS =  model_from.RNA_STCK_THETA6_TS ;
    model_to.RNA_STCK_THETA6_TC =  model_from.RNA_STCK_THETA6_TC ;
    model_to.STCK_THETAB1_A =  model_from.STCK_THETAB1_A ;
    model_to.STCK_THETAB1_B =  model_from.STCK_THETAB1_B ;
    model_to.STCK_THETAB1_T0 =  model_from.STCK_THETAB1_T0 ;
    model_to.STCK_THETAB1_TS =  model_from.STCK_THETAB1_TS ;
    model_to.STCK_THETAB1_TC =  model_from.STCK_THETAB1_TC ;
    model_to.STCK_THETAB2_A =  model_from.STCK_THETAB2_A ;
    model_to.STCK_THETAB2_B =  model_from.STCK_THETAB2_B ;
    model_to.STCK_THETAB2_T0 =  model_from.STCK_THETAB2_T0 ;
    model_to.STCK_THETAB2_TS =  model_from.STCK_THETAB2_TS ;
    model_to.STCK_THETAB2_TC =  model_from.STCK_THETAB2_TC ;
    model_to.RNA_STCK_PHI1_A =  model_from.RNA_STCK_PHI1_A ;
    model_to.RNA_STCK_PHI1_B =  model_from.RNA_STCK_PHI1_B ;
    model_to.RNA_STCK_PHI1_XC =  model_from.RNA_STCK_PHI1_XC ;
    model_to.RNA_STCK_PHI1_XS =  model_from.RNA_STCK_PHI1_XS ;
    model_to.RNA_STCK_PHI2_A =  model_from.RNA_STCK_PHI2_A ;
    model_to.RNA_STCK_PHI2_B =  model_from.RNA_STCK_PHI2_B ;
    model_to.RNA_STCK_PHI2_XC =  model_from.RNA_STCK_PHI2_XC ;
    model_to.RNA_STCK_PHI2_XS =  model_from.RNA_STCK_PHI2_XS ;
    model_to.RNA_CRST_R0 =  model_from.RNA_CRST_R0 ;
    model_to.RNA_CRST_RC =  model_from.RNA_CRST_RC ;
    model_to.RNA_CRST_K =  model_from.RNA_CRST_K ;
    model_to.RNA_CRST_BLOW =  model_from.RNA_CRST_BLOW ;
    model_to.RNA_CRST_RLOW =  model_from.RNA_CRST_RLOW ;
    model_to.RNA_CRST_RCLOW =  model_from.RNA_CRST_RCLOW ;
    model_to.RNA_CRST_BHIGH =  model_from.RNA_CRST_BHIGH ;
    model_to.RNA_CRST_RHIGH =  model_from.RNA_CRST_RHIGH ;
    model_to.RNA_CRST_RCHIGH =  model_from.RNA_CRST_RCHIGH ;
    model_to.RNA_CRST_THETA1_A =  model_from.RNA_CRST_THETA1_A ;
    model_to.RNA_CRST_THETA1_B =  model_from.RNA_CRST_THETA1_B ;
    model_to.RNA_CRST_THETA1_T0 =  model_from.RNA_CRST_THETA1_T0 ;
    model_to.RNA_CRST_THETA1_TS =  model_from.RNA_CRST_THETA1_TS ;
    model_to.RNA_CRST_THETA1_TC =  model_from.RNA_CRST_THETA1_TC ;
    model_to.RNA_CRST_THETA2_A =  model_from.RNA_CRST_THETA2_A ;
    model_to.RNA_CRST_THETA2_B =  model_from.RNA_CRST_THETA2_B ;
    model_to.RNA_CRST_THETA2_T0 =  model_from.RNA_CRST_THETA2_T0 ;
    model_to.RNA_CRST_THETA2_TS =  model_from.RNA_CRST_THETA2_TS ;
    model_to.RNA_CRST_THETA2_TC =  model_from.RNA_CRST_THETA2_TC ;
    model_to.RNA_CRST_THETA3_A =  model_from.RNA_CRST_THETA3_A ;
    model_to.RNA_CRST_THETA3_B =  model_from.RNA_CRST_THETA3_B ;
    model_to.RNA_CRST_THETA3_T0 =  model_from.RNA_CRST_THETA3_T0 ;
    model_to.RNA_CRST_THETA3_TS =  model_from.RNA_CRST_THETA3_TS ;
    model_to.RNA_CRST_THETA3_TC =  model_from.RNA_CRST_THETA3_TC ;
    model_to.RNA_CRST_THETA4_A =  model_from.RNA_CRST_THETA4_A ;
    model_to.RNA_CRST_THETA4_B =  model_from.RNA_CRST_THETA4_B ;
    model_to.RNA_CRST_THETA4_T0 =  model_from.RNA_CRST_THETA4_T0 ;
    model_to.RNA_CRST_THETA4_TS =  model_from.RNA_CRST_THETA4_TS ;
    model_to.RNA_CRST_THETA4_TC =  model_from.RNA_CRST_THETA4_TC ;
    model_to.RNA_CRST_THETA7_A =  model_from.RNA_CRST_THETA7_A ;
    model_to.RNA_CRST_THETA7_B =  model_from.RNA_CRST_THETA7_B ;
    model_to.RNA_CRST_THETA7_T0 =  model_from.RNA_CRST_THETA7_T0 ;
    model_to.RNA_CRST_THETA7_TS =  model_from.RNA_CRST_THETA7_TS ;
    model_to.RNA_CRST_THETA7_TC =  model_from.RNA_CRST_THETA7_TC ;
    model_to.RNA_CRST_THETA8_A =  model_from.RNA_CRST_THETA8_A ;
    model_to.RNA_CRST_THETA8_B =  model_from.RNA_CRST_THETA8_B ;
    model_to.RNA_CRST_THETA8_T0 =  model_from.RNA_CRST_THETA8_T0 ;
    model_to.RNA_CRST_THETA8_TS =  model_from.RNA_CRST_THETA8_TS ;
    model_to.RNA_CRST_THETA8_TC =  model_from.RNA_CRST_THETA8_TC ;
    model_to.RNA_CXST_R0 =  model_from.RNA_CXST_R0 ;
    model_to.RNA_CXST_RC =  model_from.RNA_CXST_RC ;
    model_to.RNA_CXST_K =  model_from.RNA_CXST_K ;
    model_to.RNA_CXST_BLOW =  model_from.RNA_CXST_BLOW ;
    model_to.RNA_CXST_RLOW =  model_from.RNA_CXST_RLOW ;
    model_to.RNA_CXST_RCLOW =  model_from.RNA_CXST_RCLOW ;
    model_to.RNA_CXST_BHIGH =  model_from.RNA_CXST_BHIGH ;
    model_to.RNA_CXST_RHIGH =  model_from.RNA_CXST_RHIGH ;
    model_to.RNA_CXST_RCHIGH =  model_from.RNA_CXST_RCHIGH ;
    model_to.RNA_CXST_THETA1_A =  model_from.RNA_CXST_THETA1_A ;
    model_to.RNA_CXST_THETA1_B =  model_from.RNA_CXST_THETA1_B ;
    model_to.RNA_CXST_THETA1_T0 =  model_from.RNA_CXST_THETA1_T0 ;
    model_to.RNA_CXST_THETA1_TS =  model_from.RNA_CXST_THETA1_TS ;
    model_to.RNA_CXST_THETA1_TC =  model_from.RNA_CXST_THETA1_TC ;
    model_to.RNA_CXST_THETA4_A =  model_from.RNA_CXST_THETA4_A ;
    model_to.RNA_CXST_THETA4_B =  model_from.RNA_CXST_THETA4_B ;
    model_to.RNA_CXST_THETA4_T0 =  model_from.RNA_CXST_THETA4_T0 ;
    model_to.RNA_CXST_THETA4_TS =  model_from.RNA_CXST_THETA4_TS ;
    model_to.RNA_CXST_THETA4_TC =  model_from.RNA_CXST_THETA4_TC ;
    model_to.RNA_CXST_THETA5_A =  model_from.RNA_CXST_THETA5_A ;
    model_to.RNA_CXST_THETA5_B =  model_from.RNA_CXST_THETA5_B ;
    model_to.RNA_CXST_THETA5_T0 =  model_from.RNA_CXST_THETA5_T0 ;
    model_to.RNA_CXST_THETA5_TS =  model_from.RNA_CXST_THETA5_TS ;
    model_to.RNA_CXST_THETA5_TC =  model_from.RNA_CXST_THETA5_TC ;
    model_to.RNA_CXST_THETA6_A =  model_from.RNA_CXST_THETA6_A ;
    model_to.RNA_CXST_THETA6_B =  model_from.RNA_CXST_THETA6_B ;
    model_to.RNA_CXST_THETA6_T0 =  model_from.RNA_CXST_THETA6_T0 ;
    model_to.RNA_CXST_THETA6_TS =  model_from.RNA_CXST_THETA6_TS ;
    model_to.RNA_CXST_THETA6_TC =  model_from.RNA_CXST_THETA6_TC ;
    model_to.RNA_CXST_PHI3_A =  model_from.RNA_CXST_PHI3_A ;
    model_to.RNA_CXST_PHI3_B =  model_from.RNA_CXST_PHI3_B ;
    model_to.RNA_CXST_PHI3_XC =  model_from.RNA_CXST_PHI3_XC ;
    model_to.RNA_CXST_PHI3_XS =  model_from.RNA_CXST_PHI3_XS ;
    model_to.RNA_CXST_PHI4_A =  model_from.RNA_CXST_PHI4_A ;
    model_to.RNA_CXST_PHI4_B =  model_from.RNA_CXST_PHI4_B ;
    model_to.RNA_CXST_PHI4_XC =  model_from.RNA_CXST_PHI4_XC ;
    model_to.RNA_CXST_PHI4_XS =  model_from.RNA_CXST_PHI4_XS ;

    model_to.p3_x = model_from.p3_x;
    model_to.p3_y = model_from.p3_y;
    model_to.p3_z = model_from.p3_z;

    model_to.p5_x = model_from.p5_x;
    model_to.p5_y = model_from.p5_y;
    model_to.p5_z = model_from.p5_z;

    model_to.RNA_POS_BACK_a1 = model_from.RNA_POS_BACK_a1;
    model_to.RNA_POS_BACK_a2 = model_from.RNA_POS_BACK_a2;
    model_to.RNA_POS_BACK_a3 = model_from.RNA_POS_BACK_a3;



}





template<typename number, typename number4>
CUDARNACTInteraction<number, number4>::CUDARNACTInteraction() {

    _grooving = false;
    _read_par = true;

    //Not copied over to device memory
    _spring_potential = NULL;
    _spring_eqdist = NULL;
    _affected_len = NULL;

    //Copied over to device memory
    _h_ang_params = NULL;
    _d_ang_params = NULL;

    _h_affected_indx = NULL;
    _d_affected_indx = NULL;

    _h_affected = NULL;
    _d_affected = NULL;

    _h_aff_eqdist = NULL;
    _d_aff_eqdist = NULL;

    _h_aff_gamma = NULL;
    _d_aff_gamma = NULL;

    _spring_param_size_number = 0;
    _ang_param_size = 0;
}

template<typename number, typename number4>
CUDARNACTInteraction<number, number4>::~CUDARNACTInteraction() {
    //Delete All pointers required for spring potential parameters
    if(_spring_potential != NULL) delete[] _spring_potential;
    if(_spring_eqdist != NULL) delete[] _spring_eqdist;

    if(_h_ang_params != NULL) delete[] _h_ang_params;
    if(_d_ang_params != NULL) CUDA_SAFE_CALL( hipFree(_d_ang_params) );

    if(_affected_len != NULL) delete[] _affected_len;

    if(_d_affected != NULL) CUDA_SAFE_CALL(hipFree(_d_affected));
    if(_d_aff_gamma != NULL) CUDA_SAFE_CALL(hipFree(_d_aff_gamma));
    if(_d_aff_eqdist != NULL) CUDA_SAFE_CALL(hipFree(_d_aff_eqdist));
    if(_d_affected_indx != NULL) CUDA_SAFE_CALL( hipFree(_d_affected_indx) );

    if(_h_affected != NULL) delete[] _h_affected;
    if(_h_aff_gamma != NULL) delete[] _h_aff_gamma;
    if(_h_aff_eqdist != NULL) delete[] _h_aff_eqdist;
    if(_h_affected_indx != NULL) delete[] _h_affected_indx;

}

template<typename number, typename number4>
void CUDARNACTInteraction<number, number4>::get_settings(input_file &inp) {
    _use_debye_huckel = false;
    _mismatch_repulsion = false;
    std::string inter_type;
    if (!getInputString(&inp, "parfile", this->_parameterfile, 0) == KEY_FOUND){
        throw oxDNAException("Key 'parfile' not found. Necessary for Protein sims.");

    char s[5] = "none";
    if(strcmp(this->_parameterfile, s) == 0) _read_par = false;

    }
    if (!getInputString(&inp, "topology", this->_topology_filename, 0) == KEY_FOUND){
        throw oxDNAException("Key 'topology_file' not found.");
    }

    //Function pointers constructed in the implicit RNACT Constructor Call

    _use_debye_huckel = true;
    // copy-pasted from the DNA2Interaction constructor
    _debye_huckel_half_charged_ends = true;
    this->_grooving = true;
    // end copy from DNA2Interaction

    // copied from DNA2Interaction::get_settings() (CPU), the least bad way of doing things
    getInputNumber(&inp, "salt_concentration", &_salt_concentration, 1);
    getInputBool(&inp, "dh_half_charged_ends", &_debye_huckel_half_charged_ends, 0);

    // lambda-factor (the dh length at T = 300K, I = 1.0)
    _debye_huckel_lambdafactor =  0.3667258;
    getInputFloat(&inp, "dh_lambda", &_debye_huckel_lambdafactor, 0);

    // the prefactor to the Debye-Huckel term
    _debye_huckel_prefactor = 0.0858;
    getInputFloat(&inp, "dh_strength", &_debye_huckel_prefactor, 0);
    // End copy from DNA2Interaction

    int mismatches = 0;
    if(getInputBoolAsInt(&inp, "mismatch_repulsion", &mismatches, 0) == KEY_FOUND) {
        this->_mismatch_repulsion = (bool) mismatches;
    }
    if(this->_mismatch_repulsion)
    {
        float temp;
        if(getInputFloat(&inp, "mismatch_repulsion_strength", &temp, 0) == KEY_FOUND) {
            this->_RNA_HYDR_MIS = temp;
        }
        else
        {
            this->_RNA_HYDR_MIS = 1;
        }

    }
    this->RNACTInteraction<number>::get_settings(inp);
}

template<typename number, typename number4>
void CUDARNACTInteraction<number, number4>::cuda_init(number box_side, int N) {
    CUDABaseInteraction<number, number4>::cuda_init(box_side, N);

//    Addition of Reading Parameter File -> Moved from get_settings due to needing to fill variables that are filled in the CPU version of DNACTInteraction::read_topology
    fstream top;
    int tmp1, tmp2;
    top.open(this->_topology_filename, ios::in);
    if (top.is_open()){
        top >> tmp1 >> tmp2 >> this->nrna >> this->npro >> this->nrnas;
        top >> this->_firststrand;
        top.close();
    } else {
        throw oxDNAException("Could not open Topology File");
    }


    if(this->_firststrand < 0) offset = 0;
    else if(this->_firststrand > 0) offset = this->nrna;
    else throw oxDNAException("No Strand should have an ID of 0");

    if (_read_par) {
        //Initalizing Some Host and Device Arrays for Spring Parameters
        _spring_param_size_number = sizeof(number) * (this->npro*this->npro);
        _ang_param_size = sizeof(number) * (this->npro*4);

        _spring_potential = new number[this->npro*this->npro]();
        _spring_eqdist = new number[this->npro*this->npro]();

        //Initializing Host and Device Arrays for Angular Parameters
        _h_ang_params = new number[this->npro*4]();
        CUDA_SAFE_CALL( hipMalloc(&_d_ang_params, _ang_param_size));

        char potswitch = 'x';
        number potential = 0.f, dist = 0.f;
        for(int i = 0; i< (this->npro*this->npro); i++){
            _spring_eqdist[i] = dist;
            _spring_potential[i] = potential;
            if(i < (this->npro *4 )) _h_ang_params[i] = dist;
        }

        //Checkers as Lambdas
        auto valid_angles = [](double a, double b, double c, double d)
        {
            double anglemin = min({a, b, c, d});
            double anglemax = max({a, b, c, d});
            if (anglemin < -1.0 || anglemax > 1.0){
                throw oxDNAException("Cos of Angle in Parameter File not in Valid bounds");
            }
        };

        auto valid_spring_params = [](int N, int x, int y, double d, char s, double k){
            if(x < 0 || x > N) throw oxDNAException("Invalid Particle ID %d in Parameter File", x);
            if(y < 0 || y > N) throw oxDNAException("Invalid Particle ID %d in Parameter File", y);
            if(d < 0) throw oxDNAException("Invalid Eq Distance %d in Parameter File", d);
            if(s != 's') throw oxDNAException("Potential Type %c Not Supported", s);
            if(k < 0) throw oxDNAException("Spring Constant %f Not Supported", k);
        };

        //Reading Parameter File
        int key1, key2 = 0;
        number a0, b0, c0, d0;
        string carbons;
        fstream parameters;
        parameters.open(this->_parameterfile, ios::in);
        getline (parameters,carbons);

        //total connections
        int spring_connection_num = 0;

        //allocate and declare affected_len vector
        _affected_len = new int[this->npro]();
        for(int i = 0; i < this->npro; i++) _affected_len[i] = 0;

        //Read Parameter File
        if (parameters.is_open())
        {
            while (parameters >> key1 >> key2 >> dist >> potswitch >> potential)
            {
                valid_spring_params(N, key1, key2, dist, potswitch, potential);
                spring_connection_num += 1;

                if(offset != 0) {
                    key1 -= offset;
                    key2 -= offset;
                }

                _affected_len[key1] += 1;
                _affected_len[key2] += 1;
                //potswitch is currently unused but may be later

                if (key2 - key1 == 1){
                    //Angular Parameters
                    parameters >> a0 >> b0 >> c0 >> d0;
                    valid_angles(a0, b0, c0, d0);
                    _h_ang_params[key1*4] = a0;
                    _h_ang_params[key1*4+1] = b0;
                    _h_ang_params[key1*4+2] = c0;
                    _h_ang_params[key1*4+3] = d0;

                    _spring_potential[key1*this->npro + key2] = potential;
                    _spring_eqdist[key1*this->npro + key2] = dist;

                    _spring_potential[key2*this->npro + key1] = potential;
                    _spring_eqdist[key2*this->npro + key1] = dist;

                } else {
                    _spring_potential[key1*this->npro + key2] = potential;
                    _spring_eqdist[key1*this->npro + key2] = dist;

                    _spring_potential[key2*this->npro + key1] = potential;
                    _spring_eqdist[key2*this->npro + key1] = dist;
                }
            }
            parameters.close();
        } else {
            throw oxDNAException("ParameterFile Could Not Be Opened");
        }

        //Compressed Parameter Initialization
        _h_affected_indx = new int[this->npro + 1]();
        _h_affected = new int[spring_connection_num*2]();
        _h_aff_gamma = new number[spring_connection_num*2]();
        _h_aff_eqdist = new number[spring_connection_num*2]();
        number zero = (number) 0.f;
        for(int i = 0; i < this->npro+1; i++) _h_affected_indx[i] = 0;
        for(int i = 0; i < spring_connection_num*2; i++){
            _h_affected[i] = 0;
            _h_aff_gamma[i] = zero;
            _h_aff_eqdist[i] = zero;
        }

        //Compressed Index
        int param_indx = 0;
        //For each residue
        for(int i = 0; i < this->npro; i++){
            //Fill _h_affected filtering through larger arrays filled in parameter file reading
            for(int j = i*this->npro; j < i*this->npro+this->npro; j++){
                if(_spring_eqdist[j] != 0.f){
                    //Affected List, Access is controlled with indices in _h_affected_indx
                    _h_affected[param_indx] = j % this->npro;
                    //Stored in same way for easy access, spring constants
                    _h_aff_gamma[param_indx] = _spring_potential[j];
                    //eq_distance
                    _h_aff_eqdist[param_indx] = _spring_eqdist[j];
                    param_indx += 1;
                }
            }
        }

        //Don't need Larger arrays anymore, safe to delete
        if(_spring_eqdist != NULL) delete[] _spring_eqdist;
        _spring_eqdist = NULL; //Otherwise dangling Pointer
        if(_spring_potential != NULL) delete[] _spring_potential;
        _spring_potential = NULL;

        //Allocation and Copying of Compressed Parameters
        CUDA_SAFE_CALL(hipMalloc(&_d_affected, 2 * spring_connection_num * sizeof(int)));
        CUDA_SAFE_CALL(hipMemcpy(_d_affected, _h_affected, 2 * spring_connection_num * sizeof(int), hipMemcpyHostToDevice));

        CUDA_SAFE_CALL(hipMalloc(&_d_aff_gamma, 2 * spring_connection_num * sizeof(int)));
        CUDA_SAFE_CALL(hipMemcpy(_d_aff_gamma, _h_aff_gamma, 2 * spring_connection_num * sizeof(int), hipMemcpyHostToDevice));

        CUDA_SAFE_CALL(hipMalloc(&_d_aff_eqdist, 2 * spring_connection_num * sizeof(int)));
        CUDA_SAFE_CALL(hipMemcpy(_d_aff_eqdist, _h_aff_eqdist, 2 * spring_connection_num * sizeof(int), hipMemcpyHostToDevice));

        int ind = 0;
        _h_affected_indx[0] = 0;
        //make indx access list where: _h_affected_indx[i] lower bound of i's parameters, _h_affected_indx[i+1] upper bound of i's parameters
        for(int i = 0; i < this->npro; i++){
            ind += _affected_len[i];
            _h_affected_indx[i+1] += ind;
        }

        //Don't need this anymore
        if(_affected_len != NULL) delete[] _affected_len;
        _affected_len = NULL;

        //Allocation and copying of Indice List for accessing compressed parameters
        CUDA_SAFE_CALL(hipMalloc(&_d_affected_indx, (this->npro+1)*sizeof(int)));
        CUDA_SAFE_CALL( hipMemcpy(_d_affected_indx, _h_affected_indx, (this->npro+1)*sizeof(int), hipMemcpyHostToDevice));

        //Parameters for Bending/Torsional, _h_ang_params is filled in parameter file reading
        CUDA_SAFE_CALL(hipMemcpy(_d_ang_params, _h_ang_params, _ang_param_size, hipMemcpyHostToDevice));

        //Memory Used by Parameters
        float param_memory_mb = (spring_connection_num * 2 * sizeof(int) + 2 * spring_connection_num * 2 * sizeof(number)
                + (this->npro + 1) * sizeof(int) + 4 * this->npro * sizeof(number))/SQR(1024);
        OX_LOG(Logger::LOG_INFO, "Spring Parameters Size: %.2f MB", param_memory_mb);

    } else OX_LOG(Logger::LOG_INFO, "Parfile: NONE, No protein parameters were filled");

    // Copied from CUDADNAINTERACTION
    RNAInteraction<number>::init();

    float f_copy = 1.0;//this->_hb_multiplier;
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_hb_multi), &f_copy, sizeof(float)) );

    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_N), &N, sizeof(int)) );

    //mismatch repulsion modification
    if(this->_mismatch_repulsion)
    {
        float tempmis = -1.0 * this->_RNA_HYDR_MIS / this->model->RNA_HYDR_EPS;
        this->F1_EPS[RNA_HYDR_F1][0][0] *= tempmis;
        this->F1_SHIFT[RNA_HYDR_F1][0][0] *= tempmis;
    }

    number tmp[50];
    for(int i = 0; i < 2; i++) for(int j = 0; j < 5; j++) for(int k = 0; k < 5; k++) tmp[i*25 + j*5 + k] = this->F1_EPS[i][j][k];

    COPY_ARRAY_TO_CONSTANT(MD_F1_EPS, tmp, 50);

    for(int i = 0; i < 2; i++) for(int j = 0; j < 5; j++) for(int k = 0; k < 5; k++) tmp[i*25 + j*5 + k] = this->F1_SHIFT[i][j][k];

    COPY_ARRAY_TO_CONSTANT(MD_F1_SHIFT, tmp, 50);

    COPY_ARRAY_TO_CONSTANT(MD_F1_A, this->F1_A, 2);
    COPY_ARRAY_TO_CONSTANT(MD_F1_RC, this->F1_RC, 2);
    COPY_ARRAY_TO_CONSTANT(MD_F1_R0, this->F1_R0, 2);
    COPY_ARRAY_TO_CONSTANT(MD_F1_BLOW, this->F1_BLOW, 2);
    COPY_ARRAY_TO_CONSTANT(MD_F1_BHIGH, this->F1_BHIGH, 2);
    COPY_ARRAY_TO_CONSTANT(MD_F1_RLOW, this->F1_RLOW, 2);
    COPY_ARRAY_TO_CONSTANT(MD_F1_RHIGH, this->F1_RHIGH, 2);
    COPY_ARRAY_TO_CONSTANT(MD_F1_RCLOW, this->F1_RCLOW, 2);
    COPY_ARRAY_TO_CONSTANT(MD_F1_RCHIGH, this->F1_RCHIGH, 2);

    COPY_ARRAY_TO_CONSTANT(MD_F2_K, this->F2_K, 2);
    COPY_ARRAY_TO_CONSTANT(MD_F2_RC, this->F2_RC, 2);
    COPY_ARRAY_TO_CONSTANT(MD_F2_R0, this->F2_R0, 2);
    COPY_ARRAY_TO_CONSTANT(MD_F2_BLOW, this->F2_BLOW, 2);
    COPY_ARRAY_TO_CONSTANT(MD_F2_BHIGH, this->F2_BHIGH, 2);
    COPY_ARRAY_TO_CONSTANT(MD_F2_RLOW, this->F2_RLOW, 2);
    COPY_ARRAY_TO_CONSTANT(MD_F2_RHIGH, this->F2_RHIGH, 2);
    COPY_ARRAY_TO_CONSTANT(MD_F2_RCLOW, this->F2_RCLOW, 2);
    COPY_ARRAY_TO_CONSTANT(MD_F2_RCHIGH, this->F2_RCHIGH, 2);

    COPY_ARRAY_TO_CONSTANT(MD_F5_PHI_A, this->F5_PHI_A, 4);
    COPY_ARRAY_TO_CONSTANT(MD_F5_PHI_B, this->F5_PHI_B, 4);
    COPY_ARRAY_TO_CONSTANT(MD_F5_PHI_XC, this->F5_PHI_XC, 4);
    COPY_ARRAY_TO_CONSTANT(MD_F5_PHI_XS, this->F5_PHI_XS, 4);



    CUDAModel cudamodel;
    copy_Model_to_CUDAModel_RNACT(*(this->model), cudamodel);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(rnamodel),&cudamodel,sizeof(CUDAModel))  );


    if(this->_use_edge) CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_n_forces), &this->_n_forces, sizeof(int)) );
    if (_use_debye_huckel) {
        // copied from DNA2Interaction::init() (CPU), the least bad way of doing things
        // We wish to normalise with respect to T=300K, I=1M. 300K=0.1 s.u. so divide this->_T by 0.1
        number lambda = _debye_huckel_lambdafactor * sqrt(this->_T / 0.1f) / sqrt(_salt_concentration);
        // RHIGH gives the distance at which the smoothing begins
        _debye_huckel_RHIGH = 3.0 * lambda;
        _minus_kappa = -1.0 / lambda;

        // these are just for convenience for the smoothing parameter computation
        number x = _debye_huckel_RHIGH;
        number q = _debye_huckel_prefactor;
        number l = lambda;

        // compute the some smoothing parameters
        _debye_huckel_B = -(exp(-x/l) * q * q * (x + l)*(x+l) )/(-4.*x*x*x * l * l * q );
        _debye_huckel_RC = x*(q*x + 3. * q* l )/(q * (x+l));


        number debyecut =  2. * sqrt(SQR(this->model->RNA_POS_BACK_a1)	+ SQR(this->model->RNA_POS_BACK_a2) + SQR(this->model->RNA_POS_BACK_a3)) + _debye_huckel_RC;

        // the cutoff radius for the potential should be the larger of rcut and debyecut
        if (debyecut > this->_rcut){
            this->_rcut = debyecut;
            this->_sqr_rcut = debyecut*debyecut;
        }
        // End copy from cpu interaction
        CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_RC), &_debye_huckel_RC, sizeof(float)) );
        CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_RHIGH), &_debye_huckel_RHIGH, sizeof(float)) );
        CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_prefactor), &_debye_huckel_prefactor, sizeof(float)) );
        CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_B), &_debye_huckel_B, sizeof(float)) );
        CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_minus_kappa), &_minus_kappa, sizeof(float)) );
        CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_half_charged_ends), &_debye_huckel_half_charged_ends, sizeof(bool)) );
    }
    //Constants for RNA/Protein Interaction (Same as DNA ones)
    //Backbone-Protein Excluded Volume Parameters
    _pro_backbone_sigma = 0.57f;
    _pro_backbone_rstar= 0.569f;
    _pro_backbone_b = 178699253.5f;
    _pro_backbone_rcut = 0.572934f;
    //Base-Protein Excluded Volume Parameters
    _pro_base_sigma = 0.36f;
    _pro_base_rstar= 0.359f;
    _pro_base_b = 296866090.0f;
    _pro_base_rcut = 0.362897f;
    //Protein-Protein Excluded Volume Parameters
    _pro_sigma = 0.35f;
    _pro_rstar = 0.349f;
    _pro_b = 306484596.0f;
    _pro_rcut = 0.352894;

    _kbend = this->_k_bend;
    _ktor = this->_k_tor;

    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_pro_sigma), &_pro_sigma, sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_pro_rstar), &_pro_rstar, sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_pro_rc), &_pro_rcut, sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_pro_b), &_pro_b, sizeof(float)) );

    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_pro_backbone_sigma), &_pro_backbone_sigma, sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_pro_backbone_rstar), &_pro_backbone_rstar, sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_pro_backbone_rc), &_pro_backbone_rcut, sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_pro_backbone_b), &_pro_backbone_b, sizeof(float)) );

    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_pro_base_sigma), &_pro_base_sigma, sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_pro_base_rstar), &_pro_base_rstar, sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_pro_base_rc), &_pro_base_rcut, sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_pro_base_b), &_pro_base_b, sizeof(float)) );

    //Parameters for DNACT book keeping
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(_nrna), &this->nrna, sizeof(int)) );
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(_npro), &this->npro, sizeof(int)) );
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(_offset), &this->offset, sizeof(int)) );

    //kb and kt Parameters
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(_kb), &_kbend, sizeof(float)) );
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(_kt), &_ktor, sizeof(float)) );

}

template<typename number, typename number4>
void CUDARNACTInteraction<number, number4>::compute_forces(CUDABaseList<number, number4> *lists, number4 *d_poss, GPU_quat<number> *d_orientations, number4 *d_forces, number4 *d_torques, LR_bonds *d_bonds, CUDABox<number, number4> *d_box) {
	CUDASimpleVerletList<number, number4> *_v_lists = dynamic_cast<CUDASimpleVerletList<number, number4> *>(lists);
	if(_v_lists != NULL) {
        if (_v_lists->use_edge()) {
            rnact_forces_edge_nonbonded<number, number4>
                    << < (_v_lists->_N_edges - 1) / (this->_launch_cfg.threads_per_block) + 1,
                    this->_launch_cfg.threads_per_block >> >
            (d_poss, d_orientations, this->_d_edge_forces, this->_d_edge_torques, _v_lists->_d_edge_list, _v_lists->_N_edges, d_bonds, this->_average, this->_use_debye_huckel, this->_mismatch_repulsion, d_box);

            this->_sum_edge_forces_torques(d_forces, d_torques);

            // potential for removal here
            hipDeviceSynchronize();
            CUT_CHECK_ERROR("forces_second_step error -- after non-bonded");

            rnact_forces_edge_bonded<number, number4>
                    << < this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block >> >
            (d_poss, d_orientations, d_forces, d_torques, d_bonds, this->_average, this->_use_mbf, this->_mbf_xmax, this->_mbf_finf, d_box, _d_aff_eqdist, _d_aff_gamma, _d_ang_params, _d_affected_indx, _d_affected);

        } else throw oxDNAException("Edge Approach is only implemented for DNACT Interaction using CUDA approach. Please add use_edge = 1 to your input file.");

	} else throw oxDNAException("Must Use with Lists to run simulation");
}

template class CUDARNACTInteraction<float, float4>;
template class CUDARNACTInteraction<double, LR_double4>;
